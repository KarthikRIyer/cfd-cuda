#include "hip/hip_runtime.h"
#include "jacobi.h"
#include <iostream>

__device__ double d_error;

#define MASK_DIM 3
#define MASK_OFFSET (MASK_DIM/2)

__constant__ double mask[MASK_DIM * MASK_DIM];

__global__ void jacobikernel(double *psi_d, double *psinew_d, int m, int n, int numiter) {

    // calculate each thread's global row and col
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row > 0 && row <= m && col > 0 && col <= n) {
//        for (int i = 1; i <= numiter; i++) {
//        d_error = 0;
        psinew_d[row * (m + 2) + col] =
                0.25f * (psi_d[(row - 1) * (m + 2) + col] + psi_d[(row + 1) * (m + 2) + col] +
                         psi_d[(row) * (m + 2) + col - 1] + psi_d[(row) * (m + 2) + col + 1]);

//            __syncthreads();
//            psi_d[row * (m + 2) + col] = psinew_d[row * (m + 2) + col];
//            __syncthreads();
//        }
    }
}

__global__ void convolution_2d(double *matrix, double *result, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int start_r = row - MASK_OFFSET;
    int start_c = col - MASK_OFFSET;

    double temp = 0;

    for (int i = 0; i < MASK_DIM; i++) {
        for (int j = 0; j < MASK_DIM; j++) {
            if (start_r + i >= 1 && start_r + i <= N) {
                if (start_c + j >= 1 && start_c + j <= N) {
                    temp += matrix[(start_r + i) * (N+2) + (start_c + j)] * mask[i * MASK_DIM + j];
                }
            }
        }
    }
    result[row * (N+2) + col] = temp;
}

//void jacobistep(double *psinew, double *psi, int m, int n) {
//    for (int i = 1; i <= m; i++) {
//        for (int j = 1; j <= n; j++) {
//            psinew[i * (m + 2) + j] = 0.25f * (psi[(i - 1) * (m + 2) + j] + psi[(i + 1) * (m + 2) + j] +
//                                               psi[(i) * (m + 2) + j - 1] + psi[(i) * (m + 2) + j + 1]);
//        }
//    }
//}

void jacobiiter_gpu(double *psi, int m, int n, int numiter, double &error) {

    double *psi_d;
    double *psinew_d;
    size_t bytes = sizeof(double) * (m + 2) * (n + 2);
    size_t bytes_m = sizeof(double) * 3 * 3;

    double *h_mask = new double[3 * 3];
    h_mask[0] = 0;
    h_mask[1] = 0.25;
    h_mask[2] = 0;
    h_mask[3] = 0.25;
    h_mask[4] = 0;
    h_mask[5] = 0.25;
    h_mask[6] = 0;
    h_mask[7] = 0.25;
    h_mask[8] = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

    // allocate memory on gpu
    hipMalloc(&psi_d, bytes);
    hipMalloc(&psinew_d, bytes);

    // copy data to gpu
//    hipMemcpy(psi_d, psi, bytes, hipMemcpyHostToDevice);
//    hipMemcpy(psinew_d, psinew, bytes, hipMemcpyHostToDevice);

    int THREADS = 16;
    int BLOCKS = (m + 2 + THREADS - 1) / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

//    for (int i = 0; i < (m + 2) * (n + 2); i++) {
//        std::cout << psi[i] << " ";
//    }
//    std::cout << "\n\n";
    hipMemcpy(psi_d, psi, bytes, hipMemcpyHostToDevice);
    for (int i = 1; i <= numiter; i++) {
//        jacobikernel<<<blocks, threads>>>(psi_d, psinew_d, m, n, numiter);
        convolution_2d<<<blocks, threads>>>(psi_d, psinew_d, m);
        hipMemcpy(psi_d, psinew_d, bytes, hipMemcpyDeviceToDevice);
    }

    hipMemcpy(psi, psi_d, bytes, hipMemcpyDeviceToHost);

//    for (int i = 0; i < (m + 2) * (n + 2); i++) {
//        std::cout << psi[i] << " ";
//    }
//    std::cout << "\n\n";
//
//    for (int i = 0; i<(m+2)*(n+2); i++){
//        std::cout<<psi[i]<<" ";
//    }

//    double e;
//    hipMemcpyFromSymbol(&e, HIP_SYMBOL("d_error"), sizeof(e), 0, hipMemcpyDeviceToHost);
//    error = e;

    hipFree(psi_d);
    hipFree(psinew_d);
    delete[] h_mask;
}

// parallelise
void jacobistep(double *psinew, double *psi, int m, int n) {
    for (int i = 1; i <= m; i++) {
        for (int j = 1; j <= m; j++) {
            psinew[i * (m + 2) + j] = 0.25f * (psi[(i - 1) * (m + 2) + j] + psi[(i + 1) * (m + 2) + j] +
                                               psi[(i) * (m + 2) + j - 1] + psi[(i) * (m + 2) + j + 1]);
        }
    }
}

// parallelise
double deltasq(double *newarr, double *oldarr, int m, int n) {
    double dsq = 0;
    double tmp;

    for (int i = 1; i <= m; i++) {
        for (int j = 1; j <= m; j++) {
            tmp = newarr[i * (m + 2) + j] - oldarr[i * (m + 2) + j];
            dsq += tmp * tmp;
        }
    }

    return dsq;
}